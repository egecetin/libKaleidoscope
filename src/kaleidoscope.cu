#include "hip/hip_runtime.h"
#include "kaleidoscope-cuda.hpp"

#include <cstdio>

extern "C"
{
#include "kaleidoscope.h"
}

__global__ void dimImage(double k, size_t nPoints, unsigned char *imgIn, unsigned char *imgOut)
{
	size_t offset = threadIdx.x + blockIdx.x * blockDim.x;
	if (offset < nPoints)
		*(imgOut + offset) = *(imgIn + offset) * k;
}

__global__ void transformImage(KaleidoscopeHandle *handler, unsigned char *imgIn, unsigned char *imgOut)
{
	size_t offset = threadIdx.x + blockIdx.x * blockDim.x;
	printf("%d : %x\n", offset, handler);
	if (offset >= handler->nPoints)
		return;

	TransformationInfo *infoPtr = handler->pTransferFunc + offset;
	printf("%d %d\n", infoPtr->dstOffset, infoPtr->srcOffset);
	for (int idx = 0; idx < handler->nComponents; ++idx)
		*(imgOut + infoPtr->dstOffset + idx) = *(imgIn + infoPtr->srcOffset + idx);
}

__host__ void initKaleidoscopeCuda(KaleidoscopeHandle *handlerGpu, double k, int n, int width, int height,
								   int nComponents, double scaleDown, hipStream_t &stream)
{
	if (!initKaleidoscope(handlerGpu, k, n, width, height, nComponents, scaleDown))
	{
		TransformationInfo *ptr;
		hipMallocAsync((void **)&ptr, sizeof(TransformationInfo) * handlerGpu->nPoints, stream);
		hipMemcpyAsync(ptr, handlerGpu->pTransferFunc, sizeof(TransformationInfo) * handlerGpu->nPoints,
						hipMemcpyHostToDevice, stream);
		hipStreamSynchronize(stream);

		free(handlerGpu->pTransferFunc);
		handlerGpu->pTransferFunc = ptr;

		// Calculate kernel sizes
		int minGridSize = 0;
		int blockSize = 0;

		hipOccupancyMaxPotentialBlockSize(&minGridSize, &blockSize, dimImage, 0, width * height * nComponents);
		handlerGpu->blockSizeDim = blockSize;
		handlerGpu->gridSizeDim = (width * height * nComponents + blockSize - 1) / blockSize;

		hipOccupancyMaxPotentialBlockSize(&minGridSize, &blockSize, transformImage, 0, handlerGpu->nPoints);
		handlerGpu->blockSizeTransform = blockSize;
		handlerGpu->gridSizeTransform = (handlerGpu->nPoints + blockSize - 1) / blockSize;

		return;
	}

	fprintf(stderr, "Can't initialize GPU");
	deInitKaleidoscope(handlerGpu);
}

__host__ void deInitKaleidoscopeCuda(KaleidoscopeHandle *handlerGpu, hipStream_t &stream)
{
	hipFreeAsync(handlerGpu->pTransferFunc, stream);
	hipStreamSynchronize(stream);
	handlerGpu->pTransferFunc = nullptr;
}

__host__ void processKaleidoscopeCuda(KaleidoscopeHandle *handler, unsigned char *imgIn, unsigned char *imgOut,
									  hipStream_t &stream)
{
	dimImage<<<handler->gridSizeDim, handler->blockSizeDim, 0, stream>>>(
		handler->k, handler->width * handler->height * handler->nComponents, imgIn, imgOut);
	transformImage<<<handler->gridSizeTransform, handler->blockSizeTransform, 0, stream>>>(handler, imgIn, imgOut);
	hipStreamSynchronize(stream);
}

__host__ void initDeviceMemory(unsigned char **ptr, unsigned long long siz, hipStream_t &stream)
{
	hipMallocAsync((void **)ptr, siz, stream);
	hipStreamSynchronize(stream);
}

__host__ void uploadToDeviceImageData(unsigned char *hostData, unsigned char *deviceData, unsigned long long siz,
									  hipStream_t &stream)
{
	hipMemcpyAsync(deviceData, hostData, siz, hipMemcpyHostToDevice, stream);
	hipStreamSynchronize(stream);
}

__host__ void downloadFromDeviceImageData(unsigned char *deviceData, unsigned char *hostData, unsigned long long siz,
										  hipStream_t &stream)
{
	hipMemcpyAsync(hostData, deviceData, siz, hipMemcpyDeviceToHost, stream);
	hipStreamSynchronize(stream);
}
