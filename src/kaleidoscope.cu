#include "hip/hip_runtime.h"
#include "kaleidoscope.cuh"

#include <stdexcept>

__device__ void dimImage(uint8_t *inImg, uint8_t *outImg, size_t nPoints, double k)
{
	size_t offset = threadIdx.x + blockIdx.x * blockDim.x;
	if (offset < nPoints)
		*(outImg + offset) = *(inImg + offset) * k;
}

__device__ void transformImage(uint8_t *inImg, uint8_t *outImg, int nComponents, TransformationInfo *info,
								size_t nPoints)
{
	size_t offset = threadIdx.x + blockIdx.x * blockDim.x;
	if (offset >= nPoints)
		return;

	TransformationInfo *infoPtr = info + offset;
	for (int idx = 0; idx < nComponents; ++idx)
		*(outImg + infoPtr->dstOffset + idx) = *(inImg + infoPtr->srcOffset + idx);
}

__global__ void _processImage(uint8_t *inImg, uint8_t *outImg, size_t nPixels, double dimConst, std::pair<int, int> dimSizes,
								hipStream_t stream)
{
	dimImage<<<dimSizes.second, dimSizes.first, 0, stream>>>(inImg, outImg, nPixels, dimConst);
	// transformImage<<<transformSizes.second, transformSizes.first, 0, stream>>>(inImg, outImg,
	// handler.nComponents, handler.pTransferFunc, handler.nPoints);
}

namespace kalos
{
	namespace cuda
	{
		Kaleidoscope::Kaleidoscope(int nImage, int width, int height, int nComponents, double scaleDown,
								   double dimConst, hipStream_t stream)
		{
			KaleidoscopeHandle handlerLocal;
			if (initKaleidoscope(&handlerLocal, nImage, width, height, nComponents, scaleDown) != 0)
				throw std::runtime_error("Unknown error");

			// Calculate kernel sizes
			int blockSize = 0, minGridSize = 0;
			size_t nPixel = width * height * nComponents;
			hipOccupancyMaxPotentialBlockSize(&minGridSize, &blockSize, dimImage, 0, nPixel);
			dimSizes = std::pair<int, int>(blockSize, (nPixel + blockSize - 1) / blockSize);

			hipOccupancyMaxPotentialBlockSize(&minGridSize, &blockSize, transformImage, 0, handler.nPoints);
			transformSizes = std::pair<int, int>(blockSize, (handler.nPoints + blockSize - 1) / blockSize);

			// Move transform information
			handler.width = handlerLocal.width;
			handler.height = handlerLocal.height;
			handler.nComponents = handlerLocal.nComponents;
			handler.nPoints = handlerLocal.nPoints;

			hipMallocAsync(&(handler.pTransferFunc), sizeof(TransformationInfo) * handlerLocal.nPoints, stream);
			hipMemcpyAsync(handler.pTransferFunc, handlerLocal.pTransferFunc,
							sizeof(TransformationInfo) * handlerLocal.nPoints, hipMemcpyHostToDevice, stream);

			deInitKaleidoscope(&handlerLocal);
		}

		void Kaleidoscope::processImage(uint8_t *inImg, uint8_t *outImg, size_t nPixels, double dimConst,
										hipStream_t stream)
		{
			_processImage(inImg, outImg, nPixels, dimConst, dimSizes, stream);
		}

		Kaleidoscope::~Kaleidoscope() { hipFree(handler.pTransferFunc); }

	} // namespace cuda
} // namespace kalos
