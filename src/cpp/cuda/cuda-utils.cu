#include "cuda/cuda-utils.hpp"

__host__ void initDeviceMemory(std::unique_ptr<uint8_t> &ptr, size_t siz, hipStream_t &stream)
{
	hipMallocAsync((void **)&(ptr.get()), siz, stream);
}

__host__ void deInitDeviceMemory(std::unique_ptr<uint8_t> &ptr, hipStream_t &stream)
{
	hipFreeAsync((void *)ptr.get(), stream);
}

__host__ void uploadToDeviceImageData(uint8_t *hostData, uint8_t *deviceData, size_t siz, hipStream_t &stream)
{
	hipMemcpyAsync(deviceData, hostData, siz, hipMemcpyHostToDevice, stream);
}

__host__ void downloadFromDeviceImageData(uint8_t *deviceData, uint8_t *hostData, size_t siz, hipStream_t &stream)
{
	hipMemcpyAsync(hostData, deviceData, siz, hipMemcpyDeviceToHost, stream);
}
